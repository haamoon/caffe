#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/tracking_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SwitchLayerForward(const int nthreads,
    const Dtype* const bottom_data, const Dtype* switch_data,
    Dtype* const top_data, int input_offset) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int switch_index = index / input_offset;
    
    if(switch_data[swith_index] == 1) {
    	top_data[index] = bottom_data[index];
  	} else if(switch_data[swith_index] == 0) {
  		int mat_index = index % input_offset;
  		if(mat_index % (D_2_ + 1) != 0) {
  			top_data[index] = 0;
  		} else {
  			top_data[index] = 1;
  		}
  	}
  	else {
  		LOG(FATAL) << "switch_data can be whether 0 or 1";
  	}
  }
}

template <typename Dtype>
__global__ void SwitchLayerBackward(const int nthreads,
    const Dtype* const top_diff, const Dtype* switch_data,
    Dtype* const bottom_diff, int input_offset) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int switch_index = index / input_offset;
    bottom_diff[index] = (switch_data[swith_index] == 1) ? top_diff[index] : 0;
  }
}

template <typename Dtype>
void SwitchLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  
  const Dtype* input_data = bottom[0]->gpu_data();
  const Dtype* switch_data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  SwitchLayerForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, input_data, switch_data, top_data, input_offset_);
}

template <typename Dtype>
void SwitchLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  CHECK(!propagate_down[1]) << "Can not propagate to the switch gate!";
    
	const Dtype* switch_data = bottom[1]->gpu_data();
	const Dtype* top_diff = top[0]->gpu_diff();
	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
	int count = top[0]->count();
	
	SwitchLayerBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, switch_data, bottom_diff, input_offset_);
}

}  // namespace caffe
