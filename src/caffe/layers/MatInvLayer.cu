#include "hip/hip_runtime.h"
#include <string>
#include <utility>
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/common_layers.hpp"
#include "caffe/data_layers.hpp"
#include "caffe/layer.hpp"
#include "caffe/loss_layers.hpp"
#include "caffe/neuron_layers.hpp"
#include "caffe/proto/caffe.pb.h"
#include "caffe/tracking_layers.hpp"
#include "caffe/util/math_lapack_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void AddLambdaEye(const int nthreads,
    const Dtype* const bottom_data, Dtype* top_data, Dtype lambda, int input_offset, int lda) {
  CUDA_KERNEL_LOOP(index, nthreads) {
  	int mat_index = index % input_offset;
  	top_data[index] = (mat_index % (lda + 1) == 0) ? 
  			(bottom_data[index] + lambda) : bottom_data[index];  
  }
}


template <typename Dtype>
void MatInvLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
	const vector<Blob<Dtype>*>& top) {
	Dtype* tmp_data = tmp_buffer_.mutable_gpu_data();
	const Dtype* input_data = bottom[0]->gpu_data();
	int count = bottom[0]->count();
	
	//AddLambdaEye<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
	//			count, input_data, tmp_data, (Dtype) lambda_, offset_, dim_);
	
	caffe_gpu_inverse<Dtype>(dim_, input_data, top[0]->mutable_gpu_data(), N_);
}


template <typename Dtype>
void MatInvLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	if (!propagate_down[0]) {
    	return;
  	}
	Dtype* input_diff = bottom[0]->mutable_gpu_diff();
	const Dtype* output_data = top[0]->gpu_data();
    const Dtype* output_diff = top[0]->gpu_diff();
    
    // A' = - B^\top B' B^\top
    for (int n = 0; n < N_; ++n) {
		caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, dim_,
    		dim_, dim_,
    	    (Dtype)-1., output_data + offset_ * n, output_diff + offset_ * n,
    	    (Dtype)0., tmp_buffer_.mutable_gpu_data());
    	    
    	caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, dim_,
    		dim_, dim_,
    	    (Dtype)1., tmp_buffer_.gpu_data(), output_data + offset_ * n,
    	    (Dtype)0., input_diff + offset_ * n);    		
	}	
}

INSTANTIATE_LAYER_GPU_FUNCS(MatInvLayer);

}  // namespace caffe
